#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h> 
#include <stdlib.h> 
#include <unistd.h>

#include <sys/mman.h>
#include <fcntl.h>
#include <sys/shm.h>
#include <sys/stat.h>


#include <hip/hip_runtime_api.h>

const size_t SIZE = 1 << 10; // 1 kiB of shared memory
const char* NAME = "test_shm";


__global__ void fill_kernel(int* arr)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < (SIZE / sizeof(int)))
		arr[i] = i;

}


int main(int argc, char* argv[])
{
	// Create shm
	int fd = shm_open(NAME, O_CREAT | O_RDWR, 0666);
	if (fd < 0)
	{
		fprintf(stderr, "Failed to open shared memory %s\n", NAME);
		return EXIT_FAILURE;
	}


	int res = ftruncate(fd, SIZE);
	if (res == -1)
	{
		fprintf(stderr, "Failed to resize shared memory %s\n", NAME);
		return EXIT_FAILURE;
	}


	// Map memory
	void* ptr = mmap(0, SIZE, PROT_WRITE, MAP_SHARED, fd, 0);
	

	// Pin this POSIX shared memory
	checkCudaErrors(hipHostRegister(ptr, SIZE, hipHostRegisterDefault));


	// Generate sample data from the GPU
	void* d_ptr;
	checkCudaErrors(hipHostGetDevicePointer(&d_ptr, ptr, 0));

	const size_t nThreads = 256;
	const size_t nBlocks = (SIZE / sizeof(int) + nThreads - 1) / nThreads;
	fill_kernel<<<nThreads, nBlocks>>>((int*) d_ptr);
	checkCudaErrors(hipDeviceSynchronize());
	
	printf("%lu bytes written to shm %s\n", SIZE, NAME);

	checkCudaErrors(hipHostUnregister(ptr));

	return EXIT_SUCCESS;
}
